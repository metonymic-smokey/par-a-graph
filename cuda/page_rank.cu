#include <cstring>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__
void prGPU(double *pagerank_vector,
                              double *new_pagerank, long long *vertexArray,
	                      long long *edgeArray, long long *outDegrees, double alpha,
                              double *deltaSum, long long n, double *leak) {

  long long index = blockIdx.x * blockDim.x + threadIdx.x;
  long long numthreads = blockDim.x * gridDim.x;
  
  for (long long i = index; i < n; i += numthreads) {
    double temp = pagerank_vector[i];
    double sumVal = 0;
    for (long long neighbour = vertexArray[i]; neighbour < vertexArray[i + 1]; neighbour++) {
      double num = pagerank_vector[edgeArray[neighbour]];
      long long denom = outDegrees[edgeArray[neighbour]];
      sumVal += num / denom;
    }
    new_pagerank[i] =
      (1 - alpha) / (double) n + alpha * sumVal + *leak / (double) n;
    double delta = abs(new_pagerank[i] - temp);

    atomicAdd(deltaSum, delta);
  }
}

__global__
void calcLeak(long long n, long long* outDegrees, double* pageRank, double* leak, double alpha) {
  long long index = blockIdx.x * blockDim.x + threadIdx.x;
  long long numthreads = blockDim.x * gridDim.x;

  double localLeak = 0.0;
  for (long long i = index; i < n; i += numthreads) {
    if (outDegrees[i] == 0) {
      localLeak += pageRank[i] * alpha;
    }
  }

  atomicAdd(leak, localLeak);
}
