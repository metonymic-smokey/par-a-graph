#include <cstring>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void prGPU(int *nodes, double *pagerank_vector,
                              double *new_pagerank, int *vertexArray,
                              int *edgeArray, int *outDegrees, double alpha,
                              double *deltaSum, int n, double leak) {
  int index = threadIdx.x;
  int numthreads = blockDim.x;
  for (int i = index; i < n; i += numthreads) {
    double temp = pagerank_vector[i];
    double sumVal = 0;
    for (int neighbour = vertexArray[i]; neighbour < vertexArray[i + 1];
         neighbour++) {
      sumVal += pagerank_vector[edgeArray[neighbour]] /
                double(outDegrees[edgeArray[neighbour]]);
    }
    new_pagerank[i] =
        (1 - alpha) / double(n) + alpha * sumVal + leak / double(n);
    /* printf("New pagerank = %lf\n",new_pagerank[i]); */
    double delta = abs(new_pagerank[i] - temp);
    atomicAdd(deltaSum, delta);
  }
}

extern "C" {
void pageRankCuda(int *vertexArray, int vertexArray_size, int *edgeArray,
                  int edgeArray_size, int *outDegrees, int outDegree_size,
                  double alpha, double eps) {

  FILE* fptr;
  fptr = fopen("pr_cuda_res.txt", "w");
  int n = vertexArray_size - 1;
  double *res = (double *)malloc((n + 1) * sizeof(double));
  double *pagerank_vector = (double *)malloc(n * sizeof(double));

  for (int i = 0; i < n; i++) {
    pagerank_vector[i] = 1 / double(n);
  }

  double *new_pagerank = (double *)malloc(n * sizeof(double));

  int *nodes = (int *)malloc(n * sizeof(int));
  for (int i = 0; i < n; i++) {
    nodes[i] = i;
  }

  double *delta = (double *)malloc(n * sizeof(double));
  int iters = 0;

  int *_nodes, *_vertexArray, *_edgeArray, *_outDegrees;
  hipMalloc((void **)&_nodes, n * sizeof(int));
  hipMalloc((void **)&_vertexArray, vertexArray_size * sizeof(int));
  hipMalloc((void **)&_edgeArray, edgeArray_size * sizeof(int));
  hipMalloc((void **)&_outDegrees, outDegree_size * sizeof(int));

  double *_pagerank_vector, *_new_pagerank;
  hipMalloc((void **)&_pagerank_vector, n * sizeof(double));
  hipMalloc((void **)&_new_pagerank, n * sizeof(double));

  double *_deltaSum, _leak, _alpha;
  hipMalloc((void **)&_deltaSum, sizeof(double));
  hipMalloc((void **)&_leak, sizeof(double));
  hipMalloc((void **)&_alpha, sizeof(double));

  int _n;
  hipMalloc((void **)&_n, sizeof(n));

  while (1) {
    iters++;
    double deltaSum = 0;
    double leak = 0;

    for (int i = 0; i < n; i++) {
      if (outDegrees[i] == 0) {
        leak += pagerank_vector[i];
      }
    }

    leak *= alpha;

    // memcpy go brrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrr
    hipMemcpy(_nodes, nodes, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(_vertexArray, vertexArray, vertexArray_size * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(_edgeArray, edgeArray, edgeArray_size * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(_outDegrees, outDegrees, outDegree_size * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(_pagerank_vector, pagerank_vector, n * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(_new_pagerank, new_pagerank, n * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(_deltaSum, &deltaSum, sizeof(double), hipMemcpyHostToDevice);
    
    prGPU<<<1, 256>>>(_nodes, _pagerank_vector, _new_pagerank,
                              _vertexArray, _edgeArray, _outDegrees, alpha,
                              _deltaSum, n, leak);
    hipDeviceSynchronize();

    // memcpy go brrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrrr
    hipMemcpy(nodes, _nodes, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vertexArray, _vertexArray, vertexArray_size * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(edgeArray, _edgeArray, edgeArray_size * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(outDegrees, _outDegrees, outDegree_size * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(pagerank_vector, _pagerank_vector, n * sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(new_pagerank, _new_pagerank, n * sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(&deltaSum, _deltaSum, sizeof(double), hipMemcpyDeviceToHost);
    memcpy(pagerank_vector, new_pagerank, n * sizeof(double));

    if (deltaSum < eps) {
      break;
    }
  }

  hipFree(_nodes);
  hipFree(_vertexArray);
  hipFree(_edgeArray);
  hipFree(_outDegrees);
  hipFree(_pagerank_vector);
  hipFree(_new_pagerank);
  hipFree(_deltaSum);

  double norm = 0;
  for (int i = 0; i < n; i++) {
    norm += i;
  }

  for (int i = 0; i < n; i++) {
    pagerank_vector[i] /= norm;
  }

  fprintf(fptr, "Latest pagerank is:\n");
  for (int i = 0; i < n; i++) {
    fprintf(fptr, "%lf\n", new_pagerank[i]);
  }
  fprintf(fptr, "\nIterations = \n%d\n", iters);
  fclose(fptr);
  return;
 }
}
