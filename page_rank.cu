#include "hip/hip_runtime.h"
#include <cstring>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__
void prGPU(double *pagerank_vector,
                              double *new_pagerank, long *vertexArray,
	                      long *edgeArray, long *outDegrees, double alpha,
                              double *deltaSum, int n, double *leak) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numthreads = blockDim.x * gridDim.x;
  
  for (int i = index; i < n; i += numthreads) {
    double temp = pagerank_vector[i];
    double sumVal = 0;
    for (int neighbour = vertexArray[i]; neighbour < vertexArray[i + 1]; neighbour++) {
      double num = pagerank_vector[edgeArray[neighbour]];
      double denom = double(outDegrees[edgeArray[neighbour]]);
      /* printf("num = %lf, denom = %lf\n", num, denom); */
      sumVal += num / denom;
    }
    new_pagerank[i] =
        (1 - alpha) / double(n) + alpha * sumVal + *leak / double(n);
    double delta = abs(new_pagerank[i] - temp);

    atomicAdd(deltaSum, delta);
  }
}

__global__
void calcLeak(int n, long* outDegrees, double* pageRank, double* leak, double alpha) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numthreads = blockDim.x * gridDim.x;

  double localLeak = 0.0;
  for (int i = index; i < n; i += numthreads) {
    if (outDegrees[i] == 0) {
      localLeak += pageRank[i] * alpha;
    }
  }

  atomicAdd(leak, localLeak);
}

__global__ void 